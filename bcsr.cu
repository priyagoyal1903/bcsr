/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <iostream>
#include <map>
#include <vector>
#include <cstring>
#include <tuple>

#define BLOCKSIZE 2;
const int bsize = BLOCKSIZE;
#define NR 6
#define NC 6
#define NTOT (NR*NC)

using namespace std;

struct ebsparsematrix_t
{
  size_t nr = NR;
  size_t nc = NC;
  size_t n=NTOT; // size of matrix, N x N
  std::vector<size_t> nzrow; // for each non-zero, the row / global index
  std::vector<size_t> nzcol; // for each non-zero, the col index
  std::vector<double> entry; // non-zero values for each index
};

struct block_t {
    double matrix[bsize][bsize] = {{0.0}}; // A dense 0 padded matrix of the non-zero values
    size_t row; // The starting row of the block
    size_t col; // The starting col of the block
};

struct ebbcsrmatrix_t
{
    size_t blocksize = bsize; // Size of the blocks B*B
    size_t nnzb = 0; // Number of non-zero blocks in the BCSR matrix
    std::vector<block_t> values; // The vector of blocks
    std::vector<size_t> cols;
    std::vector<size_t> block_row_ptr;
};

// Converts a COO matrix to a BCSR matrix
void convertToBCSR(ebsparsematrix_t& ebmat, ebbcsrmatrix_t& ebbcsr) {
    std::map<std::pair<int, int>, block_t> blockmap;
    for (int n = 0; n < ebmat.entry.size(); ++n)
    {
        const int i = ebmat.nzrow[n];
        const int j = ebmat.nzcol[n];
        const double e = ebmat.entry[n];

        // Calculate block starting point
        const int ib = i / ebbcsr.blocksize;
        const int jb = j / ebbcsr.blocksize;

        // Calculate where the nz should be inside the block
        const int ii = i % ebbcsr.blocksize;
        const int jj = j % ebbcsr.blocksize;

        std::pair<int, int> key = std::pair<int, int>(ib, jb);

        if (blockmap.find(key) != blockmap.end()) {
            // already in map
            blockmap.at(key).matrix[ii][jj] = e;
        }
        else {
            // not in the map already
            block_t newblock;
            newblock.row = ib;
            newblock.col = jb;
            newblock.matrix[ii][jj] = e;
            blockmap.insert({ key, newblock });
        }
    }
    std::map<std::pair<int, int>, block_t>::iterator it;

    int prev_block_id_row = -1;
    int count = 0;
    for (it = blockmap.begin(); it != blockmap.end(); it++) {
        ebbcsr.values.push_back(it->second);
        ebbcsr.cols.push_back(it->second.col*bsize);
        ebbcsr.nnzb++;
        if (it->first.first != prev_block_id_row) {
            ebbcsr.block_row_ptr.push_back(count);
            prev_block_id_row = it->first.first;
        }
        count++;
    }
    ebbcsr.block_row_ptr.push_back(count);

}

__global__ void
bcsr_kernel(int n_block_rows, int bs, size_t *col_ids, size_t *row_ptr, block_t* data, double *x, double *y)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int row = idx % bs;
  const int block_row = idx / bs;
  const int first_block = row_ptr[block_row];
  const int last_block = row_ptr[block_row + 1];
  if (row < bs && block_row < n_block_rows)
    {      
      double local_out = 0.0;
      for (int block = first_block; block < last_block; block++)
      {
        int first_col = data[block].col; 
                for (int j=0; j<bs; j++) {
                 local_out +=  (double)((double) x[first_col+j] * (double)data[block].matrix[row][j]);
                /* printf("block: %d, x: %f, mat[%d][%d] = %f,  local_out=%f\n", block, x[first_col+j], row, j, data[block].matrix[row][j], local_out);*/
                }
       }
       y[block_row*bs+row] = local_out;
      //printf("y[%d]=%f\n", block_row*bs+row, y[block_row*bs+row]);
    }
}


/**
 * Host main routine
 */
int main(void)
{
    // Launch the  CUDA Kernel
    int threadsPerBlock = 16;
    int blocksPerGrid = 16;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    vector<size_t> rows{ 0,    0,   0,   0,    0,   0,   1,  1,   1,   1,  1,   2,   2,   2,   3,   4,  4};
    vector<size_t> cols{ 0,    1,   2,   3,    4,   5,   0,  1,   2,   4,  5,  0,   3,   4,   3,   1,  2};
    vector<double> vals{ 0.7, 0.9, 0.2, 0.3,  0.4, 0.5, 0.6,0.7,  0.8, 0.5,0.1, 0.6, 0.9, 0.5, 0.4, 0.2, 0.3};
    /*Matrix 
    0.7  0.9  0.2   0.3   0.4    0.5
    06   0.7  0.8   0     0.5    0
    0.6  0    0     0.9   0.5    0
    0    0    0     0.4   0      0
    0    0.2  0.3   0     0      0
    0    0    0     0     0      0
    */
    ebsparsematrix_t mat;
    mat.nzrow = rows;
    mat.nzcol = cols;
    mat.entry = vals;
    ebbcsrmatrix_t ebbcsr;
    convertToBCSR(mat, ebbcsr);
    
    int n = mat.nr/bsize;
    //cols
    size_t *gpu_cols;
    size_t gpu_col_bytes = ebbcsr.cols.size()*sizeof(size_t);
    hipMalloc(&gpu_cols, gpu_col_bytes);
    //row_ptr
    size_t *gpu_row_ptr;
    size_t gpu_row_ptr_bytes = ebbcsr.block_row_ptr.size()*sizeof(size_t);
    hipMalloc(&gpu_row_ptr, gpu_row_ptr_bytes);
    //vals
    block_t *gpu_vals;
    size_t gpu_vals_bytes = ebbcsr.values.size()*sizeof(block_t);
    hipMalloc(&gpu_vals, gpu_vals_bytes);


    // dx dy
   // int x_size = NR; //ebbcsr.block_row_ptr.size();
   // int y_size = NR; //ebbcsr.cols.size();
    
    double *d_y;
    double *d_x;
   
    double* h_x = (double *) malloc(NR*sizeof(double));
    for (int i=0; i<NR; i++) {
         h_x[i] = 1.0; // (double) rand()/1111111111;
     }
    hipMalloc(&d_x, NR*sizeof(double));
    hipMalloc(&d_y, NR*sizeof(double)); 
 
    size_t* r = &ebbcsr.block_row_ptr[0];
    size_t* c = &ebbcsr.cols[0];
    block_t* data = &ebbcsr.values[0];     
    
    //Copy all host variables to device variables
    hipMemcpy( d_x, h_x, NR*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy( gpu_cols, c, gpu_col_bytes, hipMemcpyHostToDevice);
    hipMemcpy( gpu_row_ptr, r, gpu_row_ptr_bytes, hipMemcpyHostToDevice);   
    hipMemcpy( gpu_vals, data, gpu_vals_bytes, hipMemcpyHostToDevice);


    //Launch kernel function
   bcsr_kernel<<<blocksPerGrid,threadsPerBlock>>>(n, bsize, gpu_cols, gpu_row_ptr, gpu_vals, d_x, d_y); 
   
   //Copy output from device to host
   double* h_y = (double *) malloc(NR*sizeof(double));  
   hipMemcpy( h_y, d_y, NR*sizeof(double), hipMemcpyDeviceToHost);
   printf("output: \n");
   for (int i=0; i<NR; i++) {
        printf("y[%d] = %f\n", i, h_y[i]);
   }
   printf("Done\n");

//Free all device variables 
hipFree(d_x);
hipFree(d_y);
hipFree(gpu_cols);
hipFree(gpu_vals);
hipFree(gpu_row_ptr);

return 0;
}
